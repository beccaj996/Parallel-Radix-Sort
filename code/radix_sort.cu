//new
/***************** EXAMPLE ***********************
ArrayVals:			9, 31, 4, 18
padded ArrayVals:	09, 31, 04, 18
create histogram of size 10 for buckets 0-9
which each element initialized to 0. Use a thread
on each element of ArrayVals and increment the value
in the bucket it belongs to. This will count how many
values that belong in each bucket. In the above
example the histogram values would look like this:
HISTOGRAM:	
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9 	BUCKET
--------------------------------------
2 | 1 | 0 | 1 | 0 | 0 | 0 | 0 | 0 | 0 	VALUES COUNTER
next use an array to count the OFFSET and a copy of  that OFFSET array.
This is done by taking the element value at each index of the
histogram and adding it to the value at the previous index.
OFFSET Original:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
2 | 3 | 3 | 4 | 4 | 4 | 4 | 4 | 4 | 4
												OFFSET CHANGED IS JUST A 
												COPY OF OFFSET ORIGINAL.
OFFSET Changed:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
2 | 3 | 3 | 4 | 4 | 4 | 4 | 4 | 4 | 4
^   ^		^
|	|		|
|	|		taken from 4th index in histogram plus previous (1+3)
|	|
|	taken from second index plus the first index (1+2)
|
taken from the first index in histogram (2)
The reason we create a copy is because later, when we
want to determine how to rearange the elements, we have
to decrement the values in OFFSET so they don't overwrite
each other but we must also remember the original OFFSET
values. This will become clearer later.
As you can see the numbers that repeat occur (like index 2
and 4-9) when its corresponding index in the histogram equals 0
so the value doesn't increase.
Now we need to iterate over ArrayVals again and look at
the OFFSET changed array index it corresponds with to determine
where it goes in the list. We'll create a second temporary
list so that we don't ruin the order of the elements in the
original ArrayVals. This can be done in parallel so we can
use a thread to look at each element of ArrayVals at once.
secondList[ArrayValsSize];
we will, for example, look at the first element in ArrayVals.
Its left most digit is 0 so we will look at index 0 in the 
OFFSET changed array. We notice it has a value 2 so we can place this
number at the 2nd index of the secondList array we just created.
This would be index 1 because arrays start at 0. So whatever
number fills the OFFSET changed index we subtract 1 to determine the position
to insert into the secondList. After we input into the secondList 
we want to decrement the value in OFFSET changed so that the next number
that checks can be placed in an empty spot and not overwrite
the numbers in the same bucket. This means index 0 of the OFFSET changed
array goes from 2 to 1. We do the same thing for the other three
elements in ArrayVals. 31's first digit is a 3 so look at index 3 in 
OFFSET changed and we see that it gets placed at 4-1=3 index in the secondList.
Remember to decrement the value at OFFSET changed[3] which = 4 so it becomes 3.
continue this with the next value which is 04 which means we look at 
OFFSET changed[0], because its left most digit is 0, which has a value of 1 
because the value 2 was decremented when 09 was placed in secondList above
in line 75-78. Because the value is now 1 that means we insert 04 into 
index 1-1=0 of secondList. We finish with value 18. OFFSET changed[1] (because its
left most bit is 1) has a value of 3 so we put 18 into secondList[2] 
because 3-1 = 2. After every element has been properly inserted into secondList, 
it should now look like this:
secondList:
04, 09, 18, 31
We can see that its sorted but the computer doensn't know that.
In order to be sure its sorted we iterate through the histogram
and check to see if each value is at most 1. So if any value
in histogram is greater than 1 then we can't be sure its sorted
because we don't know which threads finished first.
So next if we find a value in histogram that is greater than 1 we
look to that index but in the original OFFSET. So histogram[0] has
a value of 2 which means we look in the original OFFSET[0] to get
the value 2. This means we are working from the ranges of
0-2 in the secondList. so we create histogram and OFFSET again.
To do this we just use a recursion and basically repeate the process 
above but now only working with elements 0 and 1 based on the range
provided. We want to do the same process as above but
on the next digit to the right. so we sort 04 and 09
by counting them into the histogram and finding the OFFSET just
like above in lines 15-30.
They will each look like this:
HISTOGRAM:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
0 | 0 | 0 | 0 | 1 | 0 | 0 | 0 | 0 | 1
OFFSET:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
0 | 0 | 0 | 0 | 1 | 1 | 1 | 1 | 1 | 2
									  
We iterate over histogram and see if any values are
greater than 1. There are none so they must all be
sorted! so we iterate over histogram and when we
get to a value that is non 0 we can point to
secondList and overwrite those numbers with the
current numbers and they will be in the correct 
order. histogram[4] is the first element with a 
non 0 value. We were given ranges 0-2 from above
(see lines 103-106) so we start at 0 and point
to secondList[0] and insert 4. Then we continue
our iteration over histogram and get to 9 as the
next non 0 element. We can point to secondList[1]
to insert 9. We are done with this part so it will
return to the previous step which is line 102 where
it will continuing iterating over its histogram
looking for values greater than 1. Refer to the
histogram displayed on line 23 as displayed here:
HISTOGRAM:	
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9 	BUCKET
--------------------------------------
2 | 1 | 0 | 1 | 0 | 0 | 0 | 0 | 0 | 0 	VALUES COUNTER
We branched off initially from histogram[0] because it 
had a value greater than 1 but now we are back and can 
continue. The rest of the elemnts contain either a 0 or 1 
so don't need to be sorted anymore. This means secondList
contains the sorted array. 
All that is left is to use threads for each element
of secondList and copy their value into the original
array ArrayVals because ArrayVals is the one that
was sent from the CPU that needs to go back to the CPU.
The array is sorted and we are done!
**************************************************/

//new


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

// #define MAX 2147483647;
#define MAX 99;

unsigned int * valuesList;
unsigned int totalNumbers;

void printArray(int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}

void printArrayU(unsigned int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}


__global__ void radixSort(unsigned int* valuesList, int digit, int arraySize, int* histogram, int* mainOffset, int* mainOffsetAfter) {

	// each element is corresponds to a bucket from 0-9
	// each element initialized to 0.
//	__shared__ int histogram[10];
	// int OFFSETOriginal[10];
	__shared__ int OFFSETChanged[10];

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	if (tid < arraySize)
	  atomicAdd(&histogram[valuesList[tid]/digit], 1);
	__syncthreads();

	// find offset values
	// OFFSETOriginal[0] = histogram[0];
	OFFSETChanged[0] = histogram[0];
//	mainHistogram[0] = histogram[0]; // for testing purposes.
	mainOffset[0] = histogram[0];
	for (int i = 1; i < 10; i++) {
//		mainHistogram[i] = histogram[i]; // for testing purposes.
		// OFFSETOriginal[i] = OFFSETOriginal[i-1] + histogram[i];
		OFFSETChanged[i] = OFFSETChanged[i-1] + histogram[i];
		mainOffset[i] = OFFSETChanged[i];
	}

	// group numbers together by bucket
	if (tid < arraySize) {
		// get the value at this instanced threads id that corresponds to the value at its index in valuesList
		int value = valuesList[tid];
		// find the max index this threads value found from valueList by looking in its offsetbucket
		int index = OFFSETChanged[valuesList[tid]/digit] - 1;
		// set every element in valuesList to 0.
		// valuesList[tid] = 0;
		// OFFSETChanged[valuesList[tid]/digit]--;
		__syncthreads();

		// place the values at their index found above as long as its empty (contains a 0)
		// if its filled from another thread already placing a value there,
		// go to the index before it and keep searching down until you find an empty spot
		
		// while (valuesList[index] != 0) {
		// 	atomicAdd(&OFFSETChanged[valuesList[tid]/digit], -1);
		// 	index = OFFSETChanged[valuesList[tid]/digit] - 1;
		// }
		
		int previousValue = value;
		valuesList[index] = value;
		atomicAdd(&OFFSETChanged[previousValue/digit], -1);
		// the list should now be sorted by the 10's digit
	}
	__syncthreads();

	for (int i = 0; i < 10; i++) {
		mainOffsetAfter[i] = OFFSETChanged[i];
	}

	return;

}

__device__ void bucketSort(int* values, int digit) {

}

int * histogram;
int * offset;
int * offsetAfter;

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);
	int histogramSize = 10;

	valuesList = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);
	histogram = (int*)malloc(sizeof(int)*histogramSize);
	offset = (int*)malloc(sizeof(int)*histogramSize);
	offsetAfter = (int*)malloc(sizeof(int)*histogramSize);
	unsigned int* d_valuesList;
	int* d_histogram;
	int* d_offset;
	int* d_offsetAfter;

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%MAX;
		// valuesList[i] = 26;
	}

	printf("VALUES BEFORE:\n");
	printArrayU(valuesList, totalNumbers);

	// fill histogram with 0's
	for (int i = 0; i < histogramSize; i++) {
		histogram[i] = 0;
		offset[i] = 0;
		offsetAfter[i] = 0;
	}

	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_histogram, sizeof(int)*histogramSize);
	hipMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_offset, sizeof(int)*histogramSize);
	hipMemcpy(d_offset, offset, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_offsetAfter, sizeof(int)*histogramSize);
	hipMemcpy(d_offsetAfter, offsetAfter, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	// digit should be the number we divide valuesList[i] by to find a particular digit.
	// i.e. if we are looking for the 10's digit we divid by 10. The 100's digit divid
	// by 100. 326 divide 100 returns 3. This example we limit our number size to only
	// be 2 digits (max_rand defined at top to be 50) so we pass in 10 as our digit to
	// find the left most digit, the 10's digit.
	// dim3 dimBlock(totalNumbers,1);
	dim3 dimGrid(totalNumbers/256 ,1, 1);
	if (totalNumbers%256) dimGrid.x++;
	dim3 dimBlock (256, 1, 1);
	int digit = 10;
	radixSort<<<(totalNumbers+255)/256, 256>>>(d_valuesList, digit, totalNumbers, d_histogram, d_offset, d_offsetAfter);

	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipFree(d_valuesList);

	hipMemcpy(histogram, d_histogram, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipFree(d_histogram);

	hipMemcpy(offset, d_offset, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipFree(d_offset);

	hipMemcpy(offsetAfter, d_offsetAfter, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipFree(d_offsetAfter);

	printf("HISTOGRAM:\n");
	printArray(histogram, histogramSize);

	printf("OFFSET BEFORE:\n");
	printArray(offset, histogramSize);

	printf("OFFSET AFTER:\n");
	printArray(offsetAfter, histogramSize);

	// print valuesList
	printf("VALUES AFTER:\n");
	printArrayU(valuesList, totalNumbers);

	return 0;
}

// Jake Manfre


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

// #define MAX 2147483647;
#define MAX 99;

unsigned int * valuesList;
unsigned int totalNumbers;

void printArray(int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}

void printArrayU(unsigned int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}


// __global__ void radixSort(unsigned int* valuesList, int digit, int arraySize, int* histogram, int* mainOffset, int* mainOffsetChanged) {

// 	 int tid = threadIdx.x + blockIdx.x * blockDim.x;

// 	// take element in values at this instanced thread and find the digit 
// 	// we're looking for thats passed in and increment the corresponding element 
// 	// in the histogram
// 	if (tid < arraySize)
// 	  atomicAdd(&histogram[valuesList[tid]/digit], 1);
// 	__syncthreads();

// 	// find offset values
// 	mainOffset[0] = histogram[0];
// 	mainOffsetChanged[0] = histogram[0];
// 	for (int i = 1; i < 10; i++) {
// 		mainOffsetChanged[i] = mainOffsetChanged[i-1] + histogram[i];
// 		mainOffset[i] = mainOffset[i-1] + histogram[i];
// 	}

// 	__shared__ int i;

// 	// group numbers together by bucket
// 	if (tid < arraySize) {

// 		int value = valuesList[tid];
// 		int index;

	
// 		for (i = 0; i < arraySize; i++) {
// 			if (tid == i) {
// 				index = mainOffsetChanged[valuesList[tid]/digit] - 1;
// 				atomicAdd(&mainOffsetChanged[valuesList[tid]/digit], -1);
// 			}
// 		}

// 		__syncthreads();

// 		valuesList[index] = value;
		
// 		/************************************************************
// 		// get the value at this instanced threads id that corresponds to the value at its index in valuesList
// 		int value = valuesList[tid];
// 		int previousValue = value;
// 		// find the max index this threads value found from valueList by looking in its offsetbucket
// 		int index = mainOffsetChanged[value/digit] - 1;

// 		__syncthreads();
		
// 		valuesList[index] = value;
// 		atomicAdd(&mainOffsetChanged[previousValue/digit], -1);
// 		// the list should now be sorted by the 10's digit

// 		*********************************************************/
// 	}
// 	__syncthreads();

// 	return;

// }

//***************************************************************************************************
//***************************************************************************************************
//***************************************************************************************************

__global__ void radix_Sort(unsigned int* valuesList, int digitMax, int digitCurrent, int startPos, int arraySize, int* histogram, int* mainOffset, int* mainOffsetChanged) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 tid += startPos;
	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	int tempDigitMax = digitMax;
	int tempDigitCurrent = digitCurrent;
	if (tid < startPos + arraySize) {
		int num = valuesList[tid];
		while (tempDigitMax != tempDigitCurrent) {
			num = valuesList[tid] / tempDigitMax;
			num *= tempDigitMax;

			tempDigitMax /= 10;
			num = valuesList[tid] - num;
		}

		atomicAdd(&histogram[num/digitCurrent], 1);
		// atomicAdd(&histogram[valuesList[tid]/digitCurrent], 1);
	}
	__syncthreads();

	// find offset before values
	mainOffset[0] = histogram[0];
	mainOffsetChanged[0] = histogram[0];
	for (int i = 1; i < 10; i++) {
		mainOffsetChanged[i] = mainOffsetChanged[i-1] + histogram[i];
		mainOffset[i] = mainOffset[i-1] + histogram[i];
	}


	__syncthreads();

	return;

}

__global__ void moveElements(unsigned int *valuesList, unsigned int *indexList, int startPos, int arraySize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	tid += startPos;

	if (tid < startPos + arraySize) {
		int val = valuesList[tid];
		int index = indexList[tid] + startPos;

		__syncthreads();

		valuesList[index] = val;
	}

	__syncthreads();

	return;

}

// int * histogram;
// int * offset;
// int * offsetAfter;
int histogramSize;
int digit;

unsigned int* d_valuesList;
// int* d_histogram;
// int* d_offset;
// int* d_offsetAfter;


void sortArray(int dig, int totalNums, int minIndex) {
	int * histogram;
	int * offset;
	int * offsetAfter;

	int* d_histogram;
	int* d_offset;
	int* d_offsetAfter;

	histogram = (int*)malloc(sizeof(int)*histogramSize);
	offset = (int*)malloc(sizeof(int)*histogramSize);
	offsetAfter = (int*)malloc(sizeof(int)*histogramSize);

	// fill histogram and offset arrays with 0's
	for (int i = 0; i < histogramSize; i++) {
		histogram[i] = 0;
		offset[i] = 0;
		offsetAfter[i] = 0;
	}

	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMalloc((void**) &d_histogram, sizeof(int)*histogramSize);
	hipMalloc((void**) &d_offset, sizeof(int)*histogramSize);
	hipMalloc((void**) &d_offsetAfter, sizeof(int)*histogramSize);

	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, hipMemcpyHostToDevice);
	hipMemcpy(d_offset, offset, sizeof(int)*histogramSize, hipMemcpyHostToDevice);
	hipMemcpy(d_offsetAfter, offsetAfter, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	radix_Sort<<<(totalNums+255)/256, 256>>>(d_valuesList, digit, dig, minIndex, totalNums, d_histogram, d_offset, d_offsetAfter);
	// radix_Sort<<<(totalNumbers+255)/256, 256>>>(d_valuesList, digit, 0, totalNumbers, d_histogram, d_offset, d_offsetAfter);

	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(histogram, d_histogram, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipMemcpy(offset, d_offset, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipMemcpy(offsetAfter, d_offsetAfter, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	// free memory on device
	hipFree(d_valuesList);
	hipFree(d_histogram);
	hipFree(d_offset);
	hipFree(d_offsetAfter);

	// find offset after values
	unsigned int *indexArray = (unsigned int*)malloc(sizeof(unsigned int)*totalNumbers);
	unsigned int *d_indexArray;
	for (int i = minIndex; i < minIndex + totalNums; i++) {
		// find the digit to sort by
		int num = valuesList[i];
		int tempDigit = digit;
		while (tempDigit != dig) {
			num = valuesList[i] / tempDigit;
			num *= tempDigit;

			tempDigit /= 10;
			num = valuesList[i] - num;
		}

		// int temp = (offsetAfter[num/dig] - 1) + minIndex;
		// indexArray[i] = temp;
		indexArray[i] = (offsetAfter[num/dig] - 1);
		// indexArray[i] += minIndex;
		offsetAfter[num/dig]--;
	}

	// copy main array and index array to device to rearrange values
	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMalloc((void **) &d_indexArray, sizeof(unsigned int)*totalNumbers);

	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_indexArray, indexArray, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);

	printf("MIN INDEX: %d\n", minIndex);
	printf("SIZE: %d\n", totalNums);
	printArrayU(indexArray, totalNumbers);
	// kernel call to rearrange the numbers in valuesList
	moveElements<<<(totalNums+255)/256,256>>>(d_valuesList, d_indexArray, minIndex, totalNums);

	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(indexArray, d_indexArray, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	// free memory
	hipFree(d_valuesList);
	hipFree(d_indexArray);

	printf("HISTOGRAM:\n");
	printArray(histogram, histogramSize);

	printf("OFFSET BEFORE:\n");
	printArray(offset, histogramSize);

	printf("OFFSET AFTER:\n");
	printArray(offsetAfter, histogramSize);

	printf("VALUES AFTER:\n");
	printArrayU(valuesList, totalNumbers);

	// call sortArray on each index of the histogram if that index value is greater than 1
	for (int i = 0; i < 10; i++) {
		if (histogram[i] > 1) {
			int minInd;
			if (i == 0) {
				minInd = 0;
			}
			else{
				minInd = offset[i-1];
			} 

			printf("RECURSION--------\n");
			sortArray(dig/10, offset[i]-minInd, minInd);
			// radix_Sort<<<(totalNums+255)/256, 256>>>(d_valuesList, digit, 0, totalNumbers, d_histogram, d_offset, d_offsetAfter);
		}
	}

	return;
}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);
	histogramSize = 10;

	valuesList = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);
	// histogram = (int*)malloc(sizeof(int)*histogramSize);
	// offset = (int*)malloc(sizeof(int)*histogramSize);
	// offsetAfter = (int*)malloc(sizeof(int)*histogramSize);

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%MAX;
	}

	printf("VALUES BEFORE:\n");
	printArrayU(valuesList, totalNumbers);

	// // fill histogram with 0's
	// for (int i = 0; i < histogramSize; i++) {
	// 	histogram[i] = 0;
	// 	offset[i] = 0;
	// 	offsetAfter[i] = 0;
	// }

	// cudaMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	// cudaMalloc((void**) &d_histogram, sizeof(int)*histogramSize);
	// cudaMalloc((void**) &d_offset, sizeof(int)*histogramSize);
	// cudaMalloc((void**) &d_offsetAfter, sizeof(int)*histogramSize);

	// cudaMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, cudaMemcpyHostToDevice);
	// cudaMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, cudaMemcpyHostToDevice);
	// cudaMemcpy(d_offset, offset, sizeof(int)*histogramSize, cudaMemcpyHostToDevice);
	// cudaMemcpy(d_offsetAfter, offsetAfter, sizeof(int)*histogramSize, cudaMemcpyHostToDevice);

	// digit should be the number we divide valuesList[i] by to find a particular digit.
	// i.e. if we are looking for the 10's digit we divid by 10. The 100's digit divid
	// by 100. 326 divide 100 returns 3. This example we limit our number size to only
	// be 2 digits (max_rand defined at top to be 50) so we pass in 10 as our digit to
	// find the left most digit, the 10's digit.

	digit = 10;
	// radixSort<<<(totalNumbers+255)/256, 256>>>(d_valuesList, digit, totalNumbers, d_histogram, d_offset, d_offsetAfter);
	sortArray(digit, totalNumbers, 0);

	// printf("HISTOGRAM:\n");
	// printArray(histogram, histogramSize);

	// printf("OFFSET BEFORE:\n");
	// printArray(offset, histogramSize);

	// printf("OFFSET AFTER:\n");
	// printArray(offsetAfter, histogramSize);

	printf("VALUES AFTER:\n");
	printArrayU(valuesList, totalNumbers);

	return 0;
}

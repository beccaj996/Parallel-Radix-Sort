#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

unsigned int * valuesList;
unsigned int totalNumbers;

__global__ unsigned int radixSort(unsigned int* values, int digit) {

	// not sure about the implenetation of the histogram part
	__shared__ histogram;
	temp_list;
	int tid = thisInstanceThread;

	// histogram contains buckets 0-9
	// use each thread to place its corresponding element of the array into 
	// the right bucket based on the current digit.
	// each recursion has its own instanced histogram.

	// iterate over each list in the histogram. Each list corresponds to a different bucket.
	// if each list has more than 1 value in it, call radixSort on that specific list(aka bucket) but
	// increment the digit.
	// if there is only 1 value then that list (bucket) is sorted. radiSort will return a sorted list as well.
	// append the list (bucket) at each index of the histogram to temp_list.
	for (int i = 0; i < histogramSize; i++) {
		if (histogram[i] size > 1) {
			temp_list += radixSort<<<numBlocks, numThreads>>>(histogram[i], digit++);
		}
		
		temp_list += histogram[i]; // append each bucket to the end of temp_list
		
	}

	return temp_list;

}

unsigned int padNumbers(unsigned int* values) {
	// pad each element with 0's to match the number with the most digits
	return paddedNumbers;
}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);

	// generate totalNumbers random numbers for valuesList

	// pad the numbers with 0's
	unsigned int paddedNumbers[totalNumbers] = paddedNumbers(valuesList);

	hipMalloc((void **) device_list, size);
	cudaMemscpy(device_list, host_list, size, hipMemcpyHostToDevice);

	// start with 10th digit. unsigned int limits the digit size to 10 so there can
	// only be a max of 10 digits.
	radixSort<<<numBlocks, numThreads>>>(paddedNumbers, 10);

	hipMemcpy(host_list, device_list,j size, hipMemcpyDeviceToHost);
	hipFree(device_list);

	// print ordered list

	return 0;
}
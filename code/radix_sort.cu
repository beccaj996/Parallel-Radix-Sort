#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

unsigned int * valuesList;
unsigned int totalNumbers;

/***************** EXAMPLE ***********************

ArrayVals: 			9, 31, 4, 18

padded arrayVals:	09, 18, 04, 31

Sort into histogram by leftMost digit (digit 1).
This can be done in parallel. The order in which
they are placed in their corresponding bucket
doesn't matter. So we can use a thread for each
element of the array and fill the buckets
simultaneously.


bucket: 	0  | 1  | 3
		   -------------
values:		09 | 18 | 31		
			04 |	|  

Once all the elements of the array have been placed
in a bucket we must synch the threads before we 
continue.

__syncthreads();

Iterate through each bucket. If a bucket contains
more than 1 value in it then split it into more
buckets based on the next digit to the right.
bucket 0 contains 04 and 09 so sort these by digit 
0 (the right most digit) to get:

bucket: 	4  |  9
		   ---------
values:	   04  |  09

Iterate through each bucket. If a bucket contains
more than 1 value in it then split into more
buckets based on the next digit to the right.
Bucket 4 and 9 each contain only 1 value.

Enter each bucket in the histogram onto a list
starting from the smallest bucket and moving up
to the largest bucket. So in this example add
04 to the list and then 09 to the list and return
this list back to the previous recursion call.
This is line (34) above where it will continue onto
the next iteration of its loop. The list that just
returned to it will be sorted as we just saw.

return list: 04,09

histogram from line 17 should now look like this
after the returned sorted list:

bucket: 	0  | 1  | 3
		   -------------
values:		04 | 18 | 31		
			09 |	| 

Each bucket in the list now contains either a
sorted list or only one element, which is also
a sorted list. Therefore we can put these values
into the original array in the sorted order by
beginning with bucket 0 and moving up to bucket 3.

arrayVals: 04, 09, 18, 31

The array is sorted!

**************************************************/




__global__ unsigned int radixSort(unsigned int* values, int digit) {

	// not sure about the implenetation of the histogram part
	__shared__ histogram;
	temp_list;
	int tid = thisInstanceThread;

	// histogram contains buckets 0-9
	// use each thread to place its corresponding element of the array into 
	// the right bucket based on the current digit.
	// each recursion has its own instanced histogram.

	// iterate over each list in the histogram. Each list corresponds to a different bucket.
	// if each list has more than 1 value in it, call radixSort on that specific list(aka bucket) but
	// increment the digit.
	// if there is only 1 value then that list (bucket) is sorted. radiSort will return a sorted list as well.
	// append the list (bucket) at each index of the histogram to temp_list.

	for (int i = 0; i < histogramSize; i++) {
		if (histogram[i] size > 1) {
			// sort the values at histogram[i] (bucket[i]) by calling radixSort on that list
			histogram[i] = radixSort<<<numBlocks, numThreads>>>(histogram[i], digit++);
		}
		
		temp_list += histogram[i]; // append each bucket to the end of temp_list
	}

	return temp_list;
}

unsigned int padNumbers(unsigned int* values) {
	// pad each element with 0's to match the number with the most digits
	return paddedNumbers;
}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);

	// generate totalNumbers random numbers for valuesList

	// pad the numbers with 0's
	unsigned int paddedNumbers[totalNumbers] = paddedNumbers(valuesList);

	hipMalloc((void **) device_list, size);
	cudaMemscpy(device_list, host_list, size, hipMemcpyHostToDevice);

	// start with 10th digit. unsigned int limits the digit size to 10 so there can
	// only be a max of 10 digits.
	radixSort<<<numBlocks, numThreads>>>(paddedNumbers, 10);

	hipMemcpy(host_list, device_list,j size, hipMemcpyDeviceToHost);
	hipFree(device_list);

	// print ordered list

	return 0;
}
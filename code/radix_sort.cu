// Jake Manfre


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define MAX 2147483647;
// #define MAX 999999;

unsigned int * valuesList;
unsigned int* d_valuesList;

unsigned int totalNumbers;
int histogramSize;
int digit = 1000000000;

void printArray(int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}

void printArrayU(unsigned int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}

__global__ void radix_Sort(unsigned int* valuesList, int digitMax, int digitCurrent, int startPos, int arraySize, int* histogram, int* mainOffset, int* mainOffsetChanged) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 tid += startPos;
	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	int tempDigitMax = digitMax;
	int tempDigitCurrent = digitCurrent;
	if (tid < startPos + arraySize) {
		int num = valuesList[tid];
		while (tempDigitMax != tempDigitCurrent) {
			num = valuesList[tid] / tempDigitMax;
			num *= tempDigitMax;

			tempDigitMax /= 10;
			num = valuesList[tid] - num;
		}

		atomicAdd(&histogram[num/digitCurrent], 1);
	}
	__syncthreads();

	// find offset before values
	mainOffset[0] = histogram[0];
	mainOffsetChanged[0] = histogram[0];
	for (int i = 1; i < 10; i++) {
		mainOffsetChanged[i] = mainOffsetChanged[i-1] + histogram[i];
		mainOffset[i] = mainOffset[i-1] + histogram[i];
	}


	__syncthreads();

	return;

}

__global__ void moveElements(unsigned int *valuesList, unsigned int *indexList, int startPos, int arraySize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	tid += startPos;

	if (tid < startPos + arraySize) {
		int val = valuesList[tid];
		int index = indexList[tid] + startPos;

		__syncthreads();

		valuesList[index] = val;
	}

	__syncthreads();

	return;

}

void sortArray(int dig, int totalNums, int minIndex, int prevMin) {
	int * histogram;
	int * offset;
	int * offsetAfter;

	int* d_histogram;
	int* d_offset;
	int* d_offsetAfter;

	histogram = (int*)malloc(sizeof(int)*histogramSize);
	offset = (int*)malloc(sizeof(int)*histogramSize);
	offsetAfter = (int*)malloc(sizeof(int)*histogramSize);

	// fill histogram and offset arrays with 0's
	for (int i = 0; i < histogramSize; i++) {
		histogram[i] = 0;
		offset[i] = 0;
		offsetAfter[i] = 0;
	}

	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMalloc((void**) &d_histogram, sizeof(int)*histogramSize);
	hipMalloc((void**) &d_offset, sizeof(int)*histogramSize);
	hipMalloc((void**) &d_offsetAfter, sizeof(int)*histogramSize);

	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, hipMemcpyHostToDevice);
	hipMemcpy(d_offset, offset, sizeof(int)*histogramSize, hipMemcpyHostToDevice);
	hipMemcpy(d_offsetAfter, offsetAfter, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	radix_Sort<<<(totalNums+255)/256, 256>>>(d_valuesList, digit, dig, minIndex, totalNums, d_histogram, d_offset, d_offsetAfter);

	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(histogram, d_histogram, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipMemcpy(offset, d_offset, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipMemcpy(offsetAfter, d_offsetAfter, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	// free memory on device
	hipFree(d_valuesList);
	hipFree(d_histogram);
	hipFree(d_offset);
	hipFree(d_offsetAfter);

	// find offset after values
	unsigned int *indexArray = (unsigned int*)malloc(sizeof(unsigned int)*totalNumbers);
	unsigned int *d_indexArray;
	for (int i = minIndex; i < minIndex + totalNums; i++) {
		// find the digit to sort by
		int num = valuesList[i];
		int tempDigit = digit;
		while (tempDigit != dig) {
			num = valuesList[i] / tempDigit;
			num *= tempDigit;

			tempDigit /= 10;
			num = valuesList[i] - num;
		}

		indexArray[i] = (offsetAfter[num/dig] - 1);
		offsetAfter[num/dig]--;
	}

	// copy main array and index array to device to rearrange values
	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMalloc((void **) &d_indexArray, sizeof(unsigned int)*totalNumbers);

	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_indexArray, indexArray, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);

	// printf("MIN INDEX: %d\n", minIndex);
	// printf("SIZE: %d\n", totalNums);
	// printf("DIGIT: %d\n", dig);
	// printArrayU(indexArray, totalNumbers);
	// kernel call to rearrange the numbers in valuesList
	moveElements<<<(totalNums+255)/256,256>>>(d_valuesList, d_indexArray, minIndex, totalNums);

	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(indexArray, d_indexArray, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	// free memory
	hipFree(d_valuesList);
	hipFree(d_indexArray);

	// printf("HISTOGRAM:\n");
	// printArray(histogram, histogramSize);

	// printf("OFFSET BEFORE:\n");
	// printArray(offset, histogramSize);

	// printf("OFFSET AFTER:\n");
	// printArray(offsetAfter, histogramSize);

	// printf("VALUES AFTER:\n");
	// printArrayU(valuesList, totalNumbers);

	// call sortArray on each index of the histogram if that index value is greater than 1
	for (int i = 0; i < 10; i++) {
		if (histogram[i] > 1 && dig != 1) {
			int minInd;
			if (i == 0) {
				minInd = 0;
			}
			else{
				minInd = offset[i-1];
			} 

			// recursion
			sortArray(dig/10, offset[i]-minInd, minInd+prevMin, minInd+prevMin);
		}
	}

	return;
}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);
	histogramSize = 10;

	valuesList = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%MAX;
	}

	printf("VALUES BEFORE:\n");
	printArrayU(valuesList, totalNumbers);

	sortArray(digit, totalNumbers, 0, 0);

	printf("VALUES AFTER:\n");
	printArrayU(valuesList, totalNumbers);

	return 0;
}

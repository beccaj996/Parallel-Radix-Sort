
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

/***************** EXAMPLE ***********************

ArrayVals:			9, 31, 4, 18

padded ArrayVals:	09, 31, 04, 18

create histogram of size 10 for buckets 0-9
which each element initialized to 0. Use a thread
on each element of ArrayVals and increment the value
in the bucket it belongs to. This will count how many
values that belong in each bucket. In the above
example the histogram values would look like this:


HISTOGRAM:	
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9 	BUCKET
--------------------------------------
2 | 1 | 0 | 1 | 0 | 0 | 0 | 0 | 0 | 0 	VALUES COUNTER

next use an array to count the OFFSET and a copy of  that OFFSET array.
This is done by taking the element value at each index of the
histogram and adding it to the value at the previous index.

OFFSET Original:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
2 | 3 | 3 | 4 | 4 | 4 | 4 | 4 | 4 | 4
												OFFSET CHANGED IS JUST A 
												COPY OF OFFSET ORIGINAL.
OFFSET Changed:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
2 | 3 | 3 | 4 | 4 | 4 | 4 | 4 | 4 | 4
^   ^		^
|	|		|
|	|		taken from 4th index in histogram plus previous (1+3)
|	|
|	taken from second index plus the first index (1+2)
|
taken from the first index in histogram (2)

The reason we create a copy is because later, when we
want to determine how to rearange the elements, we have
to decrement the values in OFFSET so they don't overwrite
each other but we must also remember the original OFFSET
values. This will become clearer later.

As you can see the numbers that repeat occur (like index 2
and 4-9) when its corresponding index in the histogram equals 0
so the value doesn't increase.

Now we need to iterate over ArrayVals again and look at
the OFFSET changed array index it corresponds with to determine
where it goes in the list. We'll create a second temporary
list so that we don't ruin the order of the elements in the
original ArrayVals. This can be done in parallel so we can
use a thread to look at each element of ArrayVals at once.

secondList[ArrayValsSize];

we will, for example, look at the first element in ArrayVals.
Its left most digit is 0 so we will look at index 0 in the 
OFFSET changed array. We notice it has a value 2 so we can place this
number at the 2nd index of the secondList array we just created.
This would be index 1 because arrays start at 0. So whatever
number fills the OFFSET changed index we subtract 1 to determine the position
to insert into the secondList. After we input into the secondlList 
we want to decrement the value in OFFSET changed so that the next number
that checks can be placed in an empty spot and not overwrite
the numbers in the same bucket. This means index 0 of the OFFSET changed
array goes from 2 to 1. We do the same thing for the other three
elements in ArrayVals. 31's first digit is a 3 so look at index 3 in 
OFFSET changed and we see that it gets placed at 4-1=3 index in the secondList.
Remember to decrement the value at OFFSET changed[3] which = 4 so it becomes 3.

continue this with the next value which is 04 which means we look at 
OFFSET changed[0], because its left most digit is 0, which has a value of 1 
because the value 2 was decremented when 09 was placed in secondList above
in line 75-78. Because the value is now 1 that means we insert 04 into 
index 1-1=0 of secondList. We finish with value 18. OFFSET changed[1] (because its
left most bit is 1) has a value of 3 so we put 18 into secondList[2] 
because 3-1 = 2. After every element has been properly inserted into secondList, 
it should now look like this:

secondList:
04, 09, 18, 31

We can see that its sorted but the computer doensn't know that.
In order to be sure its sorted we iterate through the histogram
and check to see if each value is at most 1. So if any value
in histogram is greater than 1 then we can't be sure its sorted
because we don't know which threads finished first.

So next if we find a value in histogram that is greater than 1 we
look to that index but in the original OFFSET. So histogram[0] has
a value of 2 which means we look in the original OFFSET[0] to get
the value 2. This means we are working from the ranges of
0-2 in the secondList. so we create histogram and OFFSET again.
To do this we just use a recursion and basically repeate the process 
above but now only working with elements 0 and 1 based on the range
provided. We want to do the same process as above but
on the next digit to the right. so we sort 04 and 09
by counting them into the histogram and finding the OFFSET just
like above in lines 15-30.
They will each look like this:

HISTOGRAM:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
0 | 0 | 0 | 0 | 1 | 0 | 0 | 0 | 0 | 1

OFFSET:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
0 | 0 | 0 | 0 | 1 | 1 | 1 | 1 | 1 | 2
									  

We iterate over histogram and see if any values are
greater than 1. There are none so they must all be
sorted! so we iterate over histogram and when we
get to a value that is non 0 we can point to
secondList and overwrite those numbers with the
current numbers and they will be in the correct 
order. histogram[4] is the first element with a 
non 0 value. We were given ranges 0-2 from above
(see lines 103-106) so we start at 0 and point
to secondList[0] and insert 4. Then we continue
our iteration over histogram and get to 9 as the
next non 0 element. We can point to secondList[1]
to insert 9. We are done with this part so it will
return to the previous step which is line 102 where
it will continuing iterating over its histogram
looking for values greater than 1. Refer to the
histogram displayed on line 23 as displayed here:

HISTOGRAM:	
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9 	BUCKET
--------------------------------------
2 | 1 | 0 | 1 | 0 | 0 | 0 | 0 | 0 | 0 	VALUES COUNTER

We branched off initially from histogram[0] because it 
had a value greater than 1 but now we are back and can 
continue. The rest of the elemnts contain either a 0 or 1 
so don't need to be sorted anymore. This means secondList
contains the sorted array. 

All that is left is to use threads for each element
of secondList and copy their value into the original
array ArrayVals because ArrayVals is the one that
was sent from the CPU that needs to go back to the CPU.

The array is sorted and we are done!

**************************************************/

// #define RAND_MAX 2147483647;
#define RAND_MAX 50;

unsigned int * valuesList;
unsigned int totalNumbers;


__global__ void radixSort(unsigned int* values, int digit) {

	// each element is corresponds to a bucket from 0-9
	// each element initialized to 0
	int histogram[10] = { 0 };
	int OFFSETOriginal[10] = { 0 };
	int OFFSETChanged[10] = { 0 };

	// create a second temporary list of the same size
	unsigned int* tempList;

	// int tid = threadIdx.x + blockIdx.x * blockDim.x; // FIXME: Not sure if this line is correct
	int tid = threadIdx.x; 


	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	histogram[values[tid] / digit]++;
	__syncthreads();

	// find offset values
	OFFSETOriginal[0] = histogram[0];
	OFFSETChanged[0] = OFFSETOriginal[0];
	for (int i = 1; i < 10; i++) {
		OFFSETOriginal[i] = OFFSETOriginal[i-1] + histogram[i];
		OFFSETChanged[i] = OFFSETOriginal[i];
	}

	// test printing of histogram and offset
	printf("HISTOGRAM:\n");
	for (int i = 0; i < 10; i++) {
		printf("%d, ", histogram[i]);
	}

	printf("--------------------------------------\n");

	printf("OFFSETOriginal:\n");
	for (int i = 0; i < 10; i++) {
		printf("%d, ", OFFSETOriginal[i]);
	}

	printf("--------------------------------------\n");

	printf("OFFSETChanged:\n");
	for (int i = 0; i < 10; i++) {
		printf("%d, ", OFFSETChanged[i]);
	}

	printf("--------------------------------------\n");

	return;

}

__device__ void bucketSort(int* values, int digit) {

}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);

	valuesList = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);
	unsigned int* d_valuesList;

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%RAND_MAX;
	}

	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);

	// start with 10th digit. unsigned int limits the digit size to 10 so there can
	// only be a max of 10 digits.
	radixSort<<<0, 10>>>(d_valuesList, 10);

	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipFree(d_valuesList);

	// print ordered list
	for (int i = 0; i < totalNumbers; i++) {
		printf("valuesList[%d] = %d\n", i, valuesList[i]);
	}

	return 0;
}
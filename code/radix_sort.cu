// Jake Manfre


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

// #define MAX 2147483647;
#define MAX 99;

unsigned int * valuesList;
unsigned int totalNumbers;

void printArray(int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}

void printArrayU(unsigned int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}


__global__ void radixSort(unsigned int* valuesList, int digit, int arraySize, int* histogram, int* mainOffset, int* mainOffsetChanged) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	if (tid < arraySize)
	  atomicAdd(&histogram[valuesList[tid]/digit], 1);
	__syncthreads();

	// find offset values
	mainOffset[0] = histogram[0];
	mainOffsetChanged[0] = histogram[0];
	for (int i = 1; i < 10; i++) {
		mainOffsetChanged[i] = mainOffsetChanged[i-1] + histogram[i];
		mainOffset[i] = mainOffset[i-1] + histogram[i];
	}

	__shared__ int i;

	// group numbers together by bucket
	if (tid < arraySize) {

		int value = valuesList[tid];
		int index;

	
		for (i = 0; i < arraySize; i++) {
			if (tid == i) {
				index = mainOffsetChanged[valuesList[tid]/digit] - 1;
				atomicAdd(&mainOffsetChanged[valuesList[tid]/digit], -1);
			}
		}

		__syncthreads();

		valuesList[index] = value;
		
		/************************************************************
		// get the value at this instanced threads id that corresponds to the value at its index in valuesList
		int value = valuesList[tid];
		int previousValue = value;
		// find the max index this threads value found from valueList by looking in its offsetbucket
		int index = mainOffsetChanged[value/digit] - 1;

		__syncthreads();
		
		valuesList[index] = value;
		atomicAdd(&mainOffsetChanged[previousValue/digit], -1);
		// the list should now be sorted by the 10's digit

		*********************************************************/
	}
	__syncthreads();

	// for (int i = 0; i < 10; i++) {
	// 	int min;
	// 	int max;
	// 	if (histogram[i] > 1) {
	// 		// call bucket sort on that bucket and decrement digit
	// 		if (i == 0) {
	// 			min = 0;
	// 		}
	// 		else {
	// 			min = mainOffset[i-1];
	// 		} 

	// 		max = mainOffset[i] - 1;

	// 		bucketSort<<<((max-min)+255)/256, 256>>>(valuesList, min, max, digit, digit/10);
	// 	}
	// }

	return;

}

//***************************************************************************************************
//***************************************************************************************************
//***************************************************************************************************

__global__ void radix_Sort(unsigned int* valuesList, int digit, int startPos, int arraySize, int* histogram, int* mainOffset, int* mainOffsetChanged) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 tid += startPos;
	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	if (tid < arraySize)
	  atomicAdd(&histogram[valuesList[tid]/digit], 1);
	__syncthreads();

	// find offset values
	// if (tid == 0) {
		mainOffset[0] = histogram[0];
		mainOffsetChanged[0] = histogram[0];
		for (int i = 1; i < 10; i++) {
			mainOffsetChanged[i] = mainOffsetChanged[i-1] + histogram[i];
			mainOffset[i] = mainOffset[i-1] + histogram[i];
		}
	// }
	// __syncthreads();

	// group numbers together by bucket
	if (tid < arraySize) {		
		// get the value at this instanced threads id that corresponds to the value at its index in valuesList
		int value = valuesList[tid];
		__syncthreads();
		atomicAdd(&mainOffsetChanged[value/digit], -1);
	}

	__syncthreads();

	return;

}

//***************************************************************************************************
//***************************************************************************************************
//***************************************************************************************************

__device__ void bucketSort(int* valuesList, int min, int max, int highestDigit, int currentDigit) {

	// int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// // rearange specific range of original list
	// __shared__ int tempHistogram[10];
	// __shared__ int tempOffset[10];
	// __shared__ int tempOffsetChanged[10];
	// __shared__ int range;
	// range = max-min;


	// // create histogram that counts the nubmers for each bucket
	// if (tid < range) {
	// 	int num; // value at the digit we are looking for
	// 	int value = valuesList[tid];
	// 	while (highestDigit != currentDigit) {
	// 		num = value / highestDigit;
	// 		num *= highestDigit;


	// 		highestDigit /= 10;
	// 		num = value - num;
	// 		value = num;
	// 	}

	// 	// highest digit and current digit should be the same
	// 	num /= currentDigit;
	// 	// atomicAdd(tempHistogram[num], 1);
	// 	// or this?
	// 	tempHistogram[num]++;
	// }
	// __syncthreads();

	// if (tid == 0) {
	// 	tempOffset[0] = tempHistogram[0];
	// 	for (int i = 1; i < 10; i++) {
	// 		tempOffset[i] = tempOffset[i-1] + tempHistogram[i];
	// 	}
	// }

	// __syncthreads();
}

int * histogram;
int * offset;
int * offsetAfter;
int histogramSize;

unsigned int* d_valuesList;
int* d_histogram;
int* d_offset;
int* d_offsetAfter;


void sortArray() {
	// cudaMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	// cudaMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, cudaMemcpyHostToDevice);

	// cudaMalloc((void**) &d_histogram, sizeof(int)*histogramSize);
	// cudaMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, cudaMemcpyHostToDevice);

	// cudaMalloc((void**) &d_offset, sizeof(int)*histogramSize);
	// cudaMemcpy(d_offset, offset, sizeof(int)*histogramSize, cudaMemcpyHostToDevice);

	// cudaMalloc((void**) &d_offsetAfter, sizeof(int)*histogramSize);
	// cudaMemcpy(d_offsetAfter, offsetAfter, sizeof(int)*histogramSize, cudaMemcpyHostToDevice);

	// // digit should be the number we divide valuesList[i] by to find a particular digit.
	// // i.e. if we are looking for the 10's digit we divid by 10. The 100's digit divid
	// // by 100. 326 divide 100 returns 3. This example we limit our number size to only
	// // be 2 digits (max_rand defined at top to be 50) so we pass in 10 as our digit to
	// // find the left most digit, the 10's digit.
	// // dim3 dimBlock(totalNumbers,1);
	// dim3 dimGrid(totalNumbers/256 ,1, 1);
	// if (totalNumbers%256) dimGrid.x++;
	// dim3 dimBlock (256, 1, 1);
	// int digit = 10;
	// // radixSort<<<(totalNumbers+255)/256, 256>>>(d_valuesList, digit, totalNumbers, d_histogram, d_offset, d_offsetAfter);
	// radix_Sort<<<(totalNumbers+255)/256, 256>>>(d_valuesList, digit, 0, totalNumbers, d_histogram, d_offset, d_offsetAfter);

	// cudaMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, cudaMemcpyDeviceToHost);
	// cudaFree(d_valuesList);

	// cudaMemcpy(histogram, d_histogram, sizeof(int)*histogramSize, cudaMemcpyDeviceToHost);
	// cudaFree(d_histogram);

	// cudaMemcpy(offset, d_offset, sizeof(int)*histogramSize, cudaMemcpyDeviceToHost);
	// cudaFree(d_offset);

	// cudaMemcpy(offsetAfter, d_offsetAfter, sizeof(int)*histogramSize, cudaMemcpyDeviceToHost);
	// cudaFree(d_offsetAfter);
}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);
	histogramSize = 10;

	valuesList = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);
	histogram = (int*)malloc(sizeof(int)*histogramSize);
	offset = (int*)malloc(sizeof(int)*histogramSize);
	offsetAfter = (int*)malloc(sizeof(int)*histogramSize);
	// unsigned int* d_valuesList;
	// int* d_histogram;
	// int* d_offset;
	// int* d_offsetAfter;

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%MAX;
	}

	// printf("VALUES BEFORE:\n");
	// printArrayU(valuesList, totalNumbers);

	// fill histogram with 0's
	for (int i = 0; i < histogramSize; i++) {
		histogram[i] = 0;
		offset[i] = 0;
		offsetAfter[i] = 0;
	}

	// sortArray();

	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_histogram, sizeof(int)*histogramSize);
	hipMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_offset, sizeof(int)*histogramSize);
	hipMemcpy(d_offset, offset, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_offsetAfter, sizeof(int)*histogramSize);
	hipMemcpy(d_offsetAfter, offsetAfter, sizeof(int)*histogramSize, hipMemcpyHostToDevice);

	// digit should be the number we divide valuesList[i] by to find a particular digit.
	// i.e. if we are looking for the 10's digit we divid by 10. The 100's digit divid
	// by 100. 326 divide 100 returns 3. This example we limit our number size to only
	// be 2 digits (max_rand defined at top to be 50) so we pass in 10 as our digit to
	// find the left most digit, the 10's digit.
	// dim3 dimBlock(totalNumbers,1);
	dim3 dimGrid(totalNumbers/256 ,1, 1);
	if (totalNumbers%256) dimGrid.x++;
	dim3 dimBlock (256, 1, 1);
	int digit = 10;
	// radixSort<<<(totalNumbers+255)/256, 256>>>(d_valuesList, digit, totalNumbers, d_histogram, d_offset, d_offsetAfter);
	radix_Sort<<<(totalNumbers+255)/256, 256>>>(d_valuesList, digit, 0, totalNumbers, d_histogram, d_offset, d_offsetAfter);

	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipFree(d_valuesList);

	hipMemcpy(histogram, d_histogram, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipFree(d_histogram);

	hipMemcpy(offset, d_offset, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipFree(d_offset);

	hipMemcpy(offsetAfter, d_offsetAfter, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);
	hipFree(d_offsetAfter);

	printf("HISTOGRAM:\n");
	printArray(histogram, histogramSize);

	printf("OFFSET BEFORE:\n");
	printArray(offset, histogramSize);

	printf("OFFSET AFTER:\n");
	printArray(offsetAfter, histogramSize);

	// print valuesList
	// printf("VALUES AFTER:\n");
	// printArrayU(valuesList, totalNumbers);

	return 0;
}

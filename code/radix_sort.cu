#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

unsigned int * valuesList;
unsigned int totalNumbers;

/***************** EXAMPLE ***********************

ArrayVals:			9, 31, 4, 18

padded ArrayVals:	09, 31, 04, 18

create histogram of size 10 for buckets 0-9
which each element initialized to 0. Use a thread
on each element of ArrayVals and increment the value
in the bucket it belongs to. This will count how many
values that belong in each bucket. In the above
example the histogram values would look like this:


HISTOGRAM:	
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9 	BUCKET
--------------------------------------
2 | 1 | 0 | 1 | 0 | 0 | 0 | 0 | 0 | 0 	VALUES COUNTER

next use an array to count the OFFSET and a copy of  that OFFSET array.
This is done by taking the element value at each index of the
histogram and adding it to the value at the previous index.

OFFSET Original:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
2 | 3 | 3 | 4 | 4 | 4 | 4 | 4 | 4 | 4
^	^		^									OFFSET CHANGED IS JUST A 
												COPY OF OFFSET ORIGINAL.
OFFSET Changed:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
2 | 3 | 3 | 4 | 4 | 4 | 4 | 4 | 4 | 4
^   ^		^
|	|		|
|	|		taken from 4th index in histogram plus previous (1+3)
|	|
|	taken from second index plus the first index (1+2)
|
taken from the first index in histogram (2)

The reason we create a copy is because later, when we
want to determine how to rearange the elements, we have
to decrement the values in OFFSET so they don't overwrite
each other but we must also remember the original OFFSET
values. This will become clearer later.

As you can see the numbers that repeat occur (like index 2
and 4-9) when its corresponding index in the histogram equals 0
so the value doesn't increase.

Now we need to iterate over ArrayVals again and look at
the OFFSET changed array index it corresponds with to determine
where it goes in the list. We'll create a second temporary
list so that we don't ruin the order of the elements in the
original ArrayVals. This can be done in parallel so we can
use a thread to look at each element of ArrayVals at once.

secondList[ArrayValsSize];

we will, for example, look at the first element in ArrayVals.
Its left most digit is 0 so we will look at index 0 in the 
OFFSET changed array. We notice it has a value 2 so we can place this
number at the 2nd index of the secondList array we just created.
This would be index 1 because arrays start at 0. So whatever
number fills the OFFSET changed index we subtract 1 to determine the position
to insert into the secondList. After we input into the secondlList 
we want to decrement the value in OFFSET changed so that the next number
that checks can be placed in an empty spot and not overwrite
the numbers in the same bucket. This means index 0 of the OFFSET changed
array goes from 2 to 1. We do the same thing for the other three
elements in ArrayVals. 31's first digit is a 3 so look at index 3 in 
OFFSET changed and we see that it gets placed at 4-1=3 index in the secondList.
Remember to decrement the value at OFFSET changed[3] which = 4 so it becomes 3.

continue this with the next value which is 04 which means we look at 
OFFSET changed[0], because its left most digit is 0, which has a value of 1 
because the value 2 was decremented when 09 was placed in secondList above
in line 75-78. Because the value is now 1 that means we insert 04 into 
index 1-1=0 of secondList. We finish with value 18. OFFSET changed[1] (because its
left most bit is 1) has a value of 3 so we put 18 into secondList[2] 
because 3-1 = 2. After every element has been properly inserted into secondList, 
it should now look like this:

secondList:
04, 09, 18, 31

We can see that its sorted but the computer doensn't know that.
In order to be sure its sorted we iterate through the histogram
and check to see if each value is at most 1. So if any value
in histogram is greater than 1 then we can't be sure its sorted
because we don't know which threads finished first.

So next if we find a value in histogram that is greater than 1 we
look to that index but in the original OFFSET. So histogram[0] has
a value of 2 which means we look in the original OFFSET[0] to get
the value 2. This means we are working from the ranges of
0-2 in the secondList. so we create histogram and OFFSET again.
To do this we just use a recursion and basically repeate the process 
above but now only working with elements 0 and 1 based on the range
provided. We want to do the same process as above but
on the next digit to the right. so we sort 04 and 09
by counting them into the histogram and finding the OFFSET just
like above in lines 15-30.
They will each look like this:

HISTOGRAM:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
0 | 0 | 0 | 0 | 1 | 0 | 0 | 0 | 0 | 1

OFFSET:
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9
--------------------------------------
0 | 0 | 0 | 0 | 1 | 1 | 1 | 1 | 1 | 2
				^					^  

We iterate over histogram and see if any values are
greater than 1. There are none so they must all be
sorted! so we iterate over histogram and when we
get to a value that is non 0 we can point to
secondList and overwrite those numbers with the
current numbers and they will be in the correct 
order. histogram[4] is the first element with a 
non 0 value. We were given ranges 0-2 from above
(see lines 103-106) so we start at 0 and point
to secondList[0] and insert 4. Then we continue
our iteration over histogram and get to 9 as the
next non 0 element. We can point to secondList[1]
to insert 9. We are done with this part so it will
return to the previous step which is line 102 where
it will continuing iterating over its histogram
looking for values greater than 1. Refer to the
histogram displayed on line 23 as displayed here:

HISTOGRAM:	
0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | 8 | 9 	BUCKET
--------------------------------------
2 | 1 | 0 | 1 | 0 | 0 | 0 | 0 | 0 | 0 	VALUES COUNTER

We branched off initially from histogram[0] because it 
had a value greater than 1 but now we are back and can 
continue. The rest of the elemnts contain either a 0 or 1 
so don't need to be sorted anymore. This means secondList
contains the sorted array. 

All that is left is to use threads for each element
of secondList and copy their value into the original
array ArrayVals because ArrayVals is the one that
was sent from the CPU that needs to go back to the CPU.

The array is sorted and we are done!

**************************************************/




__global__ void radixSort(unsigned int* values, int arraySize, int digit) {

	// each element is corresponds to a bucket from 0-9
	// each element initialized to 0
	int histogram[10] = { 0 };
	// create a second temporary list of the same size
	unsigned int* tempList[arraySize];

	int tid = thisInstanceThread;

	// each thread looks at left most bit and increments
	// the value in the histogram it corresponds to
	__syncthreads();


}

unsigned int padNumbers(unsigned int* values) {
	// pad each element with 0's to match the number with the most digits
	return paddedNumbers;
}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);

	// generate totalNumbers random numbers for valuesList

	// pad the numbers with 0's
	unsigned int paddedNumbers[totalNumbers] = paddedNumbers(valuesList);

	hipMalloc((void **) device_list, size);
	cudaMemscpy(device_list, host_list, size, hipMemcpyHostToDevice);

	// start with 10th digit. unsigned int limits the digit size to 10 so there can
	// only be a max of 10 digits.
	radixSort<<<numBlocks, numThreads>>>(paddedNumbers, 10);

	hipMemcpy(host_list, device_list,j size, hipMemcpyDeviceToHost);
	hipFree(device_list);

	// print ordered list

	return 0;
}
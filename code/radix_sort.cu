//new


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

// #define MAX 2147483647;
#define MAX 50;

unsigned int * valuesList;
unsigned int totalNumbers;

void printArray(int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}

void printArrayU(unsigned int * array, int size) {
	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}

  	printf("]\n");
}


__global__ void radixSort(unsigned int* valuesList, int digit, int* testHistogram) {

	// each element is corresponds to a bucket from 0-9
	// each element initialized to 0
	__shared__ int histogram[10];
	int OFFSETOriginal[10] = { 0 };
	int OFFSETChanged[10] = { 0 };

	// create a second temporary list of the same size
	// unsigned int* tempList;

	 int tid = threadIdx.x + blockIdx.x * blockDim.x; // FIXME: Not sure if this line is correct
	//int tid = threadIdx.x; 


	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	if (tid < digit)
	  // histogram[valuesList[tid] / digit]++;
	  atomicAdd(&histogram[valuesList[tid]/digit], 1);
	__syncthreads();

	// find offset values
	OFFSETOriginal[0] = histogram[0];
	OFFSETChanged[0] = OFFSETOriginal[0];
	for (int i = 1; i < 10; i++) {
		testHistogram[i] = histogram[i]++;
		OFFSETOriginal[i] = OFFSETOriginal[i-1] + histogram[i];
		OFFSETChanged[i] = OFFSETOriginal[i];
	}

	return;

}

__device__ void bucketSort(int* values, int digit) {

}

int * histogram;

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);

	valuesList = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);
	histogram = (int*)malloc(sizeof(int)*10);
	unsigned int* d_valuesList;
	int* d_histogram;

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%MAX;
	}

	// fill histogram with 0's
	for (int i = 0; i < 10; i++) {
		histogram[i] = 0;
	}

	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_histogram, sizeof(int)*10);
	hipMemcpy(d_histogram, histogram, sizeof(int)*10, hipMemcpyHostToDevice);

	// start with 10th digit. unsigned int limits the digit size to 10 so there can
	// only be a max of 10 digits.
	dim3 dimBlock(10,1);
	dim3 dimGrid(1,1);
	radixSort<<<dimGrid, dimBlock>>>(d_valuesList, 10, d_histogram);

	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipFree(d_valuesList);

	hipMemcpy(histogram, d_histogram, sizeof(int)*10, hipMemcpyDeviceToHost);
	hipFree(d_histogram);

	// print valuesList
	printf("VALUES:\n");
	printArrayU(valuesList, totalNumbers);

	printf("check.\n");
	printf("HISTOGRAM:\n");
	printArray(histogram, 10);

	return 0;
}

/*****************************************
Project 3
James Albu, Rebecca Johnson, Jacob Manfre
GPU Radix Sort Algorithm
*******************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

//#define MAX 2147483647;								//largest 32bit signed integer
 #define MAX 99;

unsigned int * valuesList;							//holds values for parallel radix sort
unsigned int * valuesList2;							//array holds values for sequential radix sort
unsigned int* d_valuesList;							//holds values for device

struct timezone Idunno;
struct timeval startTime, endTime;

float totalRunningTime = 0.00000;
unsigned int totalNumbers;							//number of data values in array
int histogramSize;
int digit = 1000000000;								//largest possible place value for 32bit signed integers

//calculates running time of the radix sort algorithm
float report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}

	return (float)(sec_diff*1.0 + usec_diff/1000000.0);
}

//sequentially sorts the radix sort algorithm on the CPU in order to compare its running time to GPU
void seqSort(unsigned int * array, int size){
  int i;
  long long semiSorted[size];
  int significantDigit = 1;
  int largestNum = 1000000000;
  // Loop until we reach the largest significant digit
  while (largestNum / significantDigit > 0)
  {  
    long long bucket[10] = { 0 };
    // Counts the number of "keys" or digits that will go into each bucket
    for (i = 0; i < size; i++)
      bucket[(array[i] / significantDigit) % 10]++;
    for (i = 1; i < 10; i++)
      bucket[i] += bucket[i - 1];   
    // Use the bucket to fill a "semiSorted" array
    for (i = size - 1; i >= 0; i--)
      semiSorted[--bucket[(array[i] / significantDigit) % 10]] = array[i];
    for (i = 0; i < size; i++)
      array[i] = semiSorted[i];
    // Move to next significant digit
    significantDigit *= 10;
    
  }
}

//function to print out arrays
void printArray(int * array, int size) {	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);}
  	printf("]\n");
}

void printArrayU(unsigned int * array, int size) {	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);
	}
  	printf("]\n");
}

//main GPU kernel
//counts the number of instances for a place value and stores in a histogram
__global__ void radix_Sort(unsigned int* valuesList, int digitMax, int digitCurrent, int startPos, int arraySize, int* histogram) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 tid += startPos;
	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	int tempDigitMax = digitMax;
	int tempDigitCurrent = digitCurrent;
	if (tid < startPos + arraySize) {
		int num = valuesList[tid];
		while (tempDigitMax != tempDigitCurrent) {
			num = valuesList[tid] / tempDigitMax;
			num *= tempDigitMax;

			tempDigitMax /= 10;
			num = valuesList[tid] - num;
		}

		atomicAdd(&histogram[num/digitCurrent], 1);
	}
	__syncthreads();
	return;

}

//rearragnes the array elements to correspond to the bucket they are placed in
__global__ void moveElements(unsigned int *valuesList, unsigned int *indexList, int startPos, int arraySize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	tid += startPos;

	if (tid < startPos + arraySize) {
		int val = valuesList[tid];
		int index = indexList[tid] + startPos;

		__syncthreads();
		valuesList[index] = val;
	}
	__syncthreads();

	return;

}

//initializing the radix sort values and memory allocation functions
void sortArray(int dig, int totalNums, int minIndex, int prevMin, int placeValue) {
	int * histogram;
	int * offset;
	int * offsetAfter;

	int* d_histogram;

	histogram = (int*)malloc(sizeof(int)*histogramSize);
	offset = (int*)malloc(sizeof(int)*histogramSize);
	offsetAfter = (int*)malloc(sizeof(int)*histogramSize);

	// fill histogram and offset arrays with 0's
	for (int i = 0; i < histogramSize; i++) {
		histogram[i] = 0;
		offset[i] = 0;
		offsetAfter[i] = 0;
	}

	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMalloc((void**) &d_histogram, sizeof(int)*histogramSize);

	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, hipMemcpyHostToDevice);
        
        gettimeofday(&startTime, &Idunno);
	radix_Sort<<<(totalNums+255)/256, 256>>>(d_valuesList, digit, dig, minIndex, totalNums, d_histogram);
	totalRunningTime = totalRunningTime + report_running_time();
	
	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(histogram, d_histogram, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);

	// free memory on device
	hipFree(d_valuesList);
	hipFree(d_histogram);

	//find offset before values
	offset[0] = histogram[0];
	offsetAfter[0] = histogram[0];
	for (int i = 1; i < 10; i++) {
	   offsetAfter[i] = offsetAfter[i-1] + histogram[i];
           offset[i] = offset[i-1] + histogram[i]; 
	}

	// find offset after values
	unsigned int *indexArray = (unsigned int*)malloc(sizeof(unsigned int)*totalNumbers);
	unsigned int *d_indexArray;
	for (int i = minIndex; i < minIndex + totalNums; i++) {
		// find the digit to sort by
		int num = valuesList[i];
		int tempDigit = digit;
		while (tempDigit != dig) {
			num = valuesList[i] / tempDigit;
			num *= tempDigit;

			tempDigit /= 10;
			num = valuesList[i] - num;
		}

		indexArray[i] = (offsetAfter[num/dig] - 1);
		offsetAfter[num/dig]--;
	}

	// copy main array and index array to device to rearrange values
	hipMalloc((void **) &d_valuesList, sizeof(unsigned int)*totalNumbers);
	hipMalloc((void **) &d_indexArray, sizeof(unsigned int)*totalNumbers);

	hipMemcpy(d_valuesList, valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_indexArray, indexArray, sizeof(unsigned int)*totalNumbers, hipMemcpyHostToDevice);

	// printf("MIN INDEX: %d\n", minIndex);
	// printf("SIZE: %d\n", totalNums);
	// printf("DIGIT: %d\n", dig);
	// printArrayU(indexArray, totalNumbers);
 	
	gettimeofday(&startTime, &Idunno);
	// kernel call to rearrange the numbers in valuesList
	moveElements<<<(totalNums+255)/256,256>>>(d_valuesList, d_indexArray, minIndex, totalNums);
	totalRunningTime = totalRunningTime + report_running_time();

	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(indexArray, d_indexArray, sizeof(unsigned int)*totalNumbers, hipMemcpyDeviceToHost);
	// free memory
	hipFree(d_valuesList);
	hipFree(d_indexArray);

	 printf("HISTOGRAM:\n");
	 printArray(histogram, histogramSize);

	 printf("OFFSET BEFORE:\n");
	 printArray(offset, histogramSize);

	 printf("OFFSET AFTER:\n");
	 printArray(offsetAfter, histogramSize);

	 printf("VALUES AFTER:\n");
	 printArrayU(valuesList, totalNumbers);
printf("----------Place value-----------: %i\n", placeValue);
	// call sortArray on each index of the histogram if that index value is greater than 1
	for (int i = 0; i < 10; i++) {
		if (histogram[i] > 1 && dig != 1) {
			int minInd;
			if (i == 0) {
				minInd = 0;
			}
			else{
				minInd = offset[i-1];
			} 

			// recursion
			sortArray(dig/10, offset[i]-minInd, minInd+prevMin, minInd+prevMin, placeValue+1);
		}
	}
	
	return;
}

int main(int argc, char **argv) {

	totalNumbers = atoi(argv[1]);
	histogramSize = 10;

	valuesList = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);
	valuesList2 = (unsigned int *)malloc(sizeof(unsigned int)*totalNumbers);

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%MAX;
	}
	for (int i = 0; i < totalNumbers; i++)
		valuesList2[i] = valuesList[i];

//	printf("VALUES BEFORE:\n");
//	printArrayU(valuesList, totalNumbers);
	printf("\nGPU running time: \n");
	sortArray(digit, totalNumbers, 0, 0, 0);
        printf("%f \n", totalRunningTime);
  
        printf("CPU running time:\n");
  	gettimeofday(&startTime, &Idunno);
  	seqSort(valuesList2, totalNumbers);
  	printf("%f \n", report_running_time());

//        printf("SeqSort: \n");
//  	printArrayU(&valuesList2[0], totalNumbers);
  
//	printf("GPU sort values:\n");
//	printArrayU(valuesList, totalNumbers);

	return 0;
}

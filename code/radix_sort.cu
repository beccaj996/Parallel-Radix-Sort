/*****************************************
Project 3
James Albu, Rebecca Johnson, Jacob Manfre
GPU Radix Sort Algorithm
*******************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

int * valuesList;							//holds values for parallel radix sort
int * valuesList2;							//array holds values for sequential radix sort
int* d_valuesList;							//holds values for device

struct timezone Idunno;
struct timeval startTime, endTime;

float totalRunningTime = 0.00000;
int totalNumbers;							//number of data values in array
int histogramSize;
int digit = 1000000000;						//largest possible place value for 32bit signed integers
int MAX;


// function to print out arrays
void printArray(int * array, int size) {	
	printf("[ ");
  	for (int i = 0; i < size; i++) {
    	printf("%d ", array[i]);}
  	printf("]\n");
}

// main GPU kernel
// counts the number of instances for a place value and stores in a histogram
__global__ void radix_Sort(int* valuesList, int digitMax, int digitCurrent, int startPos, int arraySize, int* histogram) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 tid += startPos;
	// take element in values at this instanced thread and find the digit 
	// we're looking for thats passed in and increment the corresponding element 
	// in the histogram
	int tempDigitMax = digitMax;
	int tempDigitCurrent = digitCurrent;
	if (tid < startPos + arraySize) {
		int num = valuesList[tid];
		while (tempDigitMax != tempDigitCurrent) {
			num = valuesList[tid] / tempDigitMax;
			num *= tempDigitMax;

			tempDigitMax /= 10;
			num = valuesList[tid] - num;
		}

		atomicAdd(&histogram[num/digitCurrent], 1);
	}
	__syncthreads();
	return;

}

// rearragnes the array elements to correspond to the bucket they are placed in
__global__ void moveElements(int *valuesList, int *indexList, int startPos, int arraySize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	tid += startPos;

	if (tid < startPos + arraySize) {
		int val = valuesList[tid];
		int index = indexList[tid] + startPos;

		__syncthreads();
		valuesList[index] = val;
		tid += blockDim.x * blockIdx.x;
	}
	__syncthreads();

	return;

}

// initializing the radix sort values and memory allocation functions
void sortArray(int dig, int totalNums, int minIndex, int prevMin, int placeValue) {
	int * histogram;
	int * offset;
	int * offsetAfter;

	int* d_histogram;

	histogram = (int*)malloc(sizeof(int)*histogramSize);
	offset = (int*)malloc(sizeof(int)*histogramSize);
	offsetAfter = (int*)malloc(sizeof(int)*histogramSize);

	// fill histogram and offset arrays with 0's
	for (int i = 0; i < histogramSize; i++) {
		histogram[i] = 0;
		offset[i] = 0;
		offsetAfter[i] = 0;
	}

	// copy data from host to device
	hipMalloc((void **) &d_valuesList, sizeof(int)*totalNumbers);
	hipMalloc((void**) &d_histogram, sizeof(int)*histogramSize);

	hipMemcpy(d_valuesList, valuesList, sizeof(int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_histogram, histogram, sizeof(int)*histogramSize, hipMemcpyHostToDevice);
    
    // kernel call
	radix_Sort<<<(totalNums+255)/256, 256>>>(d_valuesList, digit, dig, minIndex, totalNums, d_histogram);
	
	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(histogram, d_histogram, sizeof(int)*histogramSize, hipMemcpyDeviceToHost);

	// free memory on device
	hipFree(d_valuesList);
	hipFree(d_histogram);

	// find offset before and after values
	offset[0] = histogram[0];
	offsetAfter[0] = histogram[0];
	for (int i = 1; i < 10; i++) {
		offsetAfter[i] = offsetAfter[i-1] + histogram[i];
        offset[i] = offset[i-1] + histogram[i]; 
	}

	// find offset after values
	int *indexArray = (int*)malloc(sizeof(int)*totalNumbers);
	int *d_indexArray;
	for (int i = minIndex; i < minIndex + totalNums; i++) {
		// find the digit to sort by
		int num = valuesList[i];
		int tempDigit = digit;
		while (tempDigit != dig) {
			num = valuesList[i] / tempDigit;
			num *= tempDigit;

			tempDigit /= 10;
			num = valuesList[i] - num;
		}

		indexArray[i] = (offsetAfter[num/dig] - 1);
		offsetAfter[num/dig]--;
	}

	// copy main array and index array to device to rearrange values
	hipMalloc((void **) &d_valuesList, sizeof(int)*totalNumbers);
	hipMalloc((void **) &d_indexArray, sizeof(int)*totalNumbers);

	hipMemcpy(d_valuesList, valuesList, sizeof(int)*totalNumbers, hipMemcpyHostToDevice);
	hipMemcpy(d_indexArray, indexArray, sizeof(int)*totalNumbers, hipMemcpyHostToDevice);
 	
	// kernel call to rearrange the numbers in valuesList
	moveElements<<<(totalNums+1023)/1024,1024>>>(d_valuesList, d_indexArray, minIndex, totalNums);

	// copy data back to host from the device
	hipMemcpy(valuesList, d_valuesList, sizeof(int)*totalNumbers, hipMemcpyDeviceToHost);
	hipMemcpy(indexArray, d_indexArray, sizeof(int)*totalNumbers, hipMemcpyDeviceToHost);
	// free memory
	hipFree(d_valuesList);
	hipFree(d_indexArray);

	// printf("HISTOGRAM:\n");
	// printArray(histogram, histogramSize);

	// printf("OFFSET BEFORE:\n");
	// printArray(offset, histogramSize);

	// printf("OFFSET AFTER:\n");
	// printArray(offsetAfter, histogramSize);

	// if there is more than 1 value in any index of the histogram, then those numbers
	// need to be sorted unless the digit is 1
	for (int i = 0; i < 10; i++) {
		if (histogram[i] > 1 && dig != 1) {
			int minInd;
			if (i == 0) {
				minInd = 0;
			}
			else{
				minInd = offset[i-1];
			} 

			// recursion
			sortArray(dig/10, offset[i]-minInd, minInd+prevMin, minInd+prevMin, placeValue+1);
		}
	}
	
	return;
}

int main(int argc, char **argv) {

	// array input size
	totalNumbers = atoi(argv[1]);
	// max bit size
	if (atoi(argv[2]) > 31) {
		MAX = (int)(1 << 31);
	} else {
		MAX = (int)(1 << atoi(argv[2]));
	}
	histogramSize = 10;

	valuesList = (int *)malloc(sizeof(int)*totalNumbers);

	srand(1);	
	// generate totalNumbers random numbers for valuesList
	for (int i = 0; i < totalNumbers; i++) {
		valuesList[i] = (int) rand()%MAX;
	}

	printf("VALUES BEFORE:\n");
	printArray(valuesList, totalNumbers);
	printf("---------------------------------------------\n");

	sortArray(digit, totalNumbers, 0, 0, 0);

	printf("VALUES AFTER:\n");
	printArray(valuesList, totalNumbers);

	return 0;
}

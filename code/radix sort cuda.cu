#include "hip/hip_runtime.h"
#include <stdio.h>

int blockSize = 32;
int numBlocks = 512;

//Declaration for partition sort
__device__ void partition_by_bit(unsigned int *values, unsigned int bit);

__device__ void radix_sort(unsigned int *values)
{
    int  bit;
    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit(values, bit);
        __syncthreads();
    }
}

template<class T>
__device__ T plus_scan(T *x)
{
    unsigned int i = threadIdx.x; // id of thread executing this instance
    unsigned int n = blockDim.x;  // total number of threads in this block
    unsigned int offset;          // distance between elements to be added

    for( offset = 1; offset < n; offset *= 2) {
        T t;

        if ( i >= offset ) 
            t = x[i-offset];
        
        __syncthreads();

        if ( i >= offset ) 
            x[i] = t + x[i];      // i.e., x[i] = x[i] + x[i-1]

        __syncthreads();
    }
    return x[i];
}

__device__ void partition_by_bit(unsigned int *values, unsigned int bit)
{
    unsigned int i = threadIdx.x;
    unsigned int size = blockDim.x;
    unsigned int x_i = values[i];          // value of integer at position i
    unsigned int p_i = (x_i >> bit) & 1;   // value of bit at position bit

    // Replace values array so that values[i] is the value of bit bit in
    // element i.
    values[i] = p_i;  

    // Wait for all threads to finish this.
    __syncthreads();

    // Now the values array consists of 0's and 1's, such that values[i] = 0
    // if the bit at position bit in element i was 0 and 1 otherwise.

    // Compute number of True bits (1-bits) up to and including values[i], 
    // transforming values[] so that values[i] contains the sum of the 1-bits
    // from values[0] .. values[i]
    unsigned int T_before = plus_scan(values);
/*
    plus_scan(values) returns the total number of 1-bits for all j such that
    j <= i. This is assigned to T_before, the number of 1-bits before i 
    (includes i itself)
*/

    // The plus_scan() function does not return here until all threads have
    // reached the __syncthreads() call in the last iteration of its loop
    // Therefore, when it does return, we know that the entire array has had
    // the prefix sums computed, and that values[size-1] is the sum of all
    // elements in the array, which happens to be the number of 1-bits in 
    // the current bit position.
    unsigned int T_total  = values[size-1];
    // T_total, after the scan, is the total number of 1-bits in the entire array.

    unsigned int F_total  = size - T_total;
/*    
    F_total is the total size of the array less the number of 1-bits and hence
    is the number of 0-bits.
*/
    __syncthreads();

    if ( p_i )
        values[T_before-1 + F_total] = x_i;
    else
        values[i - T_before] = x_i;

}

int main(){
	//FIXME: add arugment handler
	int numElements = 4;
	unsigned int valueArray[numElements];

	//FOR TESTING ----
	valueArray[0] = 15;
	valueArray[1] = 1;
	valueArray[2] = 8;
	valueArray[3] = 4;

	unsigned int dArray = NULL;
	hipMalloc((unsigned int)%dArray,sizeof(unsigned int) * numElements);

	hipMemcpy(dArray,valueArray,sizeof(unsigned int) * numElements, CudaMemcpyHostToDevice);

	radix_sort<<<numBlocks, blockSize>>>(dArray);

	hipMemcpy(valueArray,dArray,sizeof(unsigned int) * numElements, CudaMemcpyDeviceToHost);

	hipFree(dArray);

	for(int i = 0; i < numElements;i++){
		printf("%d \n");
	}

	return 0;
}
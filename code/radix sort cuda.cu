
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define MAX 99;

int blockSize = 256;
int numBlocks;
int* valueArray;
int * dArray;

//Declaration for partition sort
__device__ void partition_by_bit(int *values, int bit);

__global__ void radix_sort(int *values)
{

    int  bit;
    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit(values, bit);
        __syncthreads();
    }
    
}

template<class T>
__device__ T plus_scan(T *x)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // int i = threadIdx.x; // id of thread executing this instance
    int n = blockDim.x;  // total number of threads in this block
    // int n = 10;
    int offset;          // distance between elements to be added

    if (i < n) {
        for( offset = 1; offset < n; offset *= 2) {
            T t;

            if ( i >= offset ) 
                t = x[i-offset];
            
            __syncthreads();

            if ( i >= offset ) 
                x[i] = t + x[i];      // i.e., x[i] = x[i] + x[i-1]

            __syncthreads();
        }

    }
    return x[i];
}

__device__ void partition_by_bit(int *values, int bit)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // int i = threadIdx.x;
    int size = blockDim.x;
    // int size = 10;
    if (i < size) {

        int x_i = values[i];          // value of integer at position i
        int p_i = (x_i >> bit) & 1;   // value of bit at position bit

        // Replace values array so that values[i] is the value of bit bit in
        // element i.
        values[i] = p_i;  

        // Wait for all threads to finish this.
        __syncthreads();

        // Now the values array consists of 0's and 1's, such that values[i] = 0
        // if the bit at position bit in element i was 0 and 1 otherwise.

        // Compute number of True bits (1-bits) up to and including values[i], 
        // transforming values[] so that values[i] contains the sum of the 1-bits
        // from values[0] .. values[i]
        int T_before = plus_scan(values);
    /*
        plus_scan(values) returns the total number of 1-bits for all j such that
        j <= i. This is assigned to T_before, the number of 1-bits before i 
        (includes i itself)
    */

        // The plus_scan() function does not return here until all threads have
        // reached the __syncthreads() call in the last iteration of its loop
        // Therefore, when it does return, we know that the entire array has had
        // the prefix sums computed, and that values[size-1] is the sum of all
        // elements in the array, which happens to be the number of 1-bits in 
        // the current bit position.
        int T_total  = values[size-1];
        // T_total, after the scan, is the total number of 1-bits in the entire array.

        int F_total  = size - T_total;
    /*    
        F_total is the total size of the array less the number of 1-bits and hence
        is the number of 0-bits.
    */
        __syncthreads();

        if ( p_i )
            values[T_before-1 + F_total] = x_i;
        else
            values[i - T_before] = x_i;


    }

}

int main(int argc, char **argv){
	//FIXME: add arugment handler
	int numElements = atoi(argv[1]);
    numBlocks = numElements;
	// valueArray[numElements];

	valueArray = (int *)malloc(sizeof(int)*numElements);

    for (int i = 0; i < numElements; i++) {
        valueArray[i] = (int) rand()%MAX;
    }

    printf("PRINTING BEFORE:\n");
    for(int i = 0; i < numElements;i++){
        printf("%d, ",valueArray[i]);
    }

	hipMalloc((void **) &dArray, sizeof(int) * numElements);

	hipMemcpy(dArray,valueArray, sizeof(int) * numElements, hipMemcpyHostToDevice);

    // radix_sort<<<ceil(numElements/256), 256>>>(dArray);
	radix_sort<<<1, numElements>>>(dArray);

	hipMemcpy(valueArray, dArray, sizeof(int) * numElements, hipMemcpyDeviceToHost);

	hipFree(dArray);

    printf("\n\nPRINTING AFTER:\n");
	for(int i = 0; i < numElements;i++){
		printf("%d, ",valueArray[i]);
	}

    printf("DONE!\n");

	return 0;
}
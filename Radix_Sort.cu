//**********************************************************************************************************
//  Project 3
// Rebecca Johnson, James Albu, Jacob Manfre
//
// GPU Radix Sort algortihm
//
//*********************************************************************************************************


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

long long  num_dataPts;
int MAX_VALUE = 2147483647;
int num_buckets;
int blockSize = 32;
int numBlocks = 512;
struct timezone Idunno;	
struct timeval startTime, endTime;


typedef struct hist_entry{
 long long digit_count;
} bucket;

bucket *histogram;

void printArray(int * array, int size){
  
  int i;
  printf("[ ");
  for (i = 0; i < size; i++)
    printf("%d ", array[i]);
  printf("]\n");
}

__global__
void Kernel(long long *bucket, int *array, int sigDig);

void sort(int *semiSorted, int *array, bucket *histo, int size)
{  int significantDigit = 1;
   int largestNum = 1000000000;
  while (largestNum/significantDigit > 0)
  {   long long  bucket[10] = {0};
    //  long long bin[10] = {0};
      Kernel<<<numBlocks, blockSize>>>(bucket, array, significantDigit); 
      hipMemcpy(bucket, bucket, 10*sizeof(long long), hipMemcpyDeviceToHost);
    //  cudaMemcpy(array, array, size*sizeof(int), cudaMemcpyDeviceToHost);      
 //     printArray(&array[0], size); 
  //  for (int k = 1; k < 10; k++)
    //     bucket[k] += bucket[k-1];
    // for (int k = size - 1; k >= 0; k--)
     //  semiSorted[--bucket[(array[k] / significantDigit) % 10]] = array[k];
     //for (int k = 0; k < size; k++)
     //  array[k] = semiSorted[k];
     significantDigit *= 10;
}
}
__global__
 void Kernel(long long *bucket, int *array, int sigDig)
{   int k = threadIdx.x + blockDim.x;
 //   int n = blockDim.x;
//    int  semiSorted[size] = {0};
    //int significantDigit = 1;
    //int largestNum = 1000000000;

 //   while(largestNum/significantDigit > 0)
//    {
//	long long bucket[10] = {0};
//	for (int k = 0; k < size; k++)
         bucket[(array[k] /sigDig)%10] ++;
	__syncthreads();
//	for (int k = 1; k < 10; k++)
//           bucket[k] += bucket[k-1];
//	for (int k = size - 1; k >= 0; k--)
//	   semiSorted[--bucket[(array[k] / significantDigit) % 10]] = array[k];
//	for (int k = 0; k < size; k++)
//	   array[k] = semiSorted[k];
//	significantDigit *= 10;
//	__syncthreads();
  //  }
}//end of kernel

// Radix Sort

void radixSort(int * array, int size){
  int i;
  long long semiSorted[size];
  int significantDigit = 1;
  int largestNum = 1000000000;
  // Loop until we reach the largest significant digit
  while (largestNum / significantDigit > 0)
  {  
    long long bucket[10] = { 0 };
    // Counts the number of "keys" or digits that will go into each bucket
    for (i = 0; i < size; i++)
      bucket[(array[i] / significantDigit) % 10]++;
    for (i = 1; i < 10; i++)
      bucket[i] += bucket[i - 1];   
    // Use the bucket to fill a "semiSorted" array
    for (i = size - 1; i >= 0; i--)
      semiSorted[--bucket[(array[i] / significantDigit) % 10]] = array[i];
    for (i = 0; i < size; i++)
      array[i] = semiSorted[i];
    // Move to next significant digit
    significantDigit *= 10;
    
  }
}

//output the histogram
void outputHistogram()
{  int i;
   long long total_cnt = 0;
   for (i = 0; i < num_buckets; i++)
   {  if (i%10 == 0)
        printf("\n%02d: ",i);
      printf("%15lld ", histogram[i].digit_count);
      total_cnt += histogram[i].digit_count;
      if (i == num_buckets-1)
	printf("\n Total: %lld \n", total_cnt);
      else printf("| ");
   }
}

double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

int main(int argc, char **argv){
 
  bucket *dhistogram;
  num_dataPts = atoi(argv[1]);		//amount of data to sort
  num_buckets = 100;
  int data[num_dataPts]; 
  int *device_data, sortedData[num_dataPts], semiSorted[num_dataPts];
  
  hipMemset(dhistogram, 0, sizeof(bucket)*num_buckets);
  histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
  hipMalloc(&dhistogram, sizeof(bucket)*num_buckets);
  hipMalloc(&device_data, sizeof(int)*num_dataPts);

  //generate random 32 bit signed integers until we have data[num_dataPts] filled
  srand(1);
  for (int i = 0; i < num_dataPts; i++)
  { if ((int)rand() < MAX_VALUE)
      data[i] = (int)rand();
  }

  printf("Unsorted data: ");
  printArray(&data[0], num_dataPts);  
  hipMemcpy(device_data, data, num_dataPts*sizeof(int), hipMemcpyHostToDevice);

  gettimeofday(&startTime, &Idunno);
   sort(semiSorted, device_data, dhistogram, num_dataPts);
  // Kernel<<<numBlocks, blockSize>>>(semiSorted, device_data, dhistogram, num_dataPts); 
// radixSort(&data[0], num_dataPts);		//sort data using radix sort algorithm MSD
  report_running_time();
  
  hipMemcpy(histogram, dhistogram, num_buckets*sizeof(bucket), hipMemcpyDeviceToHost);
  hipMemcpy(sortedData, device_data, num_dataPts*sizeof(int), hipMemcpyDeviceToHost); 
//  printf("\nSorted List:");
//  printArray(&data[0], num_dataPts);
    printf("\nSorted data: ");
    printArray(&sortedData[0], num_dataPts);
 //   printArray(&device_data[0], num_dataPts); 
// outputHistogram();
  printf("\n");
  
  return 0;
}

